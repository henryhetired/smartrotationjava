#include "hip/hip_runtime.h"
extern "C"
__global__ void calc_entropy_atomic(float *float_image_in, float *entropy_out, int* blk) {
	//TODO: CHECK INDEX FOR ENTROPY_OUT
	//calculate entropy of a block through a single thread
        int blk_size = &blk;
	__shared__ float sum;
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		sum = 0.0;
	}
	__syncthreads();
	__shared__ float c;
	int blocksize = &blk_size*&blk_size;
	//vertical offset to get to beginning of own block
	int v_offset_to_blkrow = gridDim.x*blockDim.x*blockDim.y*blockIdx.y;
	int v_offset_to_pixrow = blockDim.x*gridDim.x*threadIdx.y;
	int h_offset = blockDim.x*blockIdx.x + threadIdx.x;
	int idx = v_offset_to_blkrow + v_offset_to_pixrow + h_offset; //idx of top left corner of the block
	int out_idx = blockIdx.y*gridDim.x + blockIdx.x;
	//normalize image
	float_image_in[idx] = float_image_in[idx] * float_image_in[idx] / (blocksize);
	atomicAdd(&sum, float_image_in[idx]);
	__syncthreads();
	__shared__ float entropy;
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		entropy = 0.0;
	}
	__syncthreads();
	float_image_in[idx] = float_image_in[idx] / sum;
	//shannon entropy
	atomicAdd(&entropy, -float_image_in[idx] * log2(float_image_in[idx]));
	__syncthreads();
	//printf("%f\n", sum2);
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		entropy_out[out_idx] = entropy;
	}
}
__global__ void thread_dct_h(float *float_image_in, float *coefficients, float *float_image_out, int* blk) {
	//dct on rows
	//summation using Kahan algorithm, very important!
  int blk_size = &blk;
	float sum = 0.0;
	float c = 0.0;
	for (int i = 0; i<blk_size; i++) {
		//printf("executing %d th task",i);

		//printf("param1 =  %d,param2 = %d,param3 = %d \n",gridDim.x*blockIdx.y*blockDim.x*blockDim.y+threadIdx.y*gridDim.x*blockDim.x+threadIdx.x+blockIdx.x*blockDim.x,threadIdx.y*blk_size+i,gridDim.x*blockIdx.y*blockDim.x*blockDim.y+threadIdx.y*gridDim.x*blockDim.x+threadIdx.x+blockIdx.x*blockDim.x+i);
		float temp = coefficients[threadIdx.y*blk_size + i] * float_image_in[gridDim.x*blockIdx.y*blockDim.x*blockDim.y + threadIdx.y*gridDim.x*blockDim.x + threadIdx.x + blockIdx.x*blockDim.x + i] - c;
		float t = sum + temp;
		c = (t - sum) - temp;
		sum = t;
	}
	float_image_out[gridDim.x*blockIdx.y*blockDim.x*blockDim.y + threadIdx.y*gridDim.x*blockDim.x + threadIdx.x + blockIdx.x*blockDim.x] = sum;
}
__global__ void thread_dct_v(float *float_image_in, float *coefficients, float *float_image_out, int* blk) {
	// dct on columns
	//summation using Kahan algorithm, very important!
  int blk_size = &blk;
	float sum = 0.0;
	float c = 0.0;
	for (int i = 0; i<blk_size; i++) {
		//printf("executing %d th task",i);

		//printf("param1 =  %d,param2 = %d,param3 = %d \n",gridDim.x*blockIdx.y*blockDim.x*blockDim.y+threadIdx.y*gridDim.x*blockDim.x+threadIdx.x+blockIdx.x*blockDim.x,threadIdx.y*blk_size+i,gridDim.x*blockIdx.y*blockDim.x*blockDim.y+threadIdx.y*gridDim.x*blockDim.x+threadIdx.x+blockIdx.x*blockDim.x+i);
		float temp = coefficients[threadIdx.y*blk_size + i] * float_image_in[gridDim.x*blockIdx.y*blockDim.x*blockDim.y + i*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x] - c;
		float t = sum + temp;
		c = (t - sum) - temp;
		sum = t;
	}
	float_image_out[gridDim.x*blockIdx.y*blockDim.x*blockDim.y + threadIdx.y*gridDim.x*blockDim.x + threadIdx.x + blockIdx.x*blockDim.x] = sum;
}
